#include "hip/hip_runtime.h"

#include "ngpu_nest_interface.h"

// C++ includes:
#include <limits>

// Includes from libnestutil:
#include "dict_util.h"

// Includes from nestkernel:
#include "exceptions.h"
#include "kernel_manager.h"
#include "universal_data_logger_impl.h"

// Includes from sli:
#include "dict.h"
#include "dictutils.h"
#include "doubledatum.h"
#include "integerdatum.h"


#include "model_manager.h"
#include "model_manager_impl.h"
#include "genericmodel.h"
#include "genericmodel_impl.h"
#include "spike_buffer.h"

#include "neurongpu_C.h"

/*
extern "C" {
  int NeuronGPU_Simulate();
  int NeuronGPU_SimulationStep();
  int NeuronGPU_SetSimTime(float sim_time);
  int NeuronGPU_PushSpikesToNodes(int n_spikes, int *node_id);
  int NeuronGPU_SetVerbosityLevel(int verbosity_level);
  int NeuronGPU_GetExtNeuronInputSpikes(int *n_spikes, int **node, int **port,
					float **spike_height,
					int include_zeros);

}
*/
  
namespace nest // template specialization must be placed in namespace
{

  ngpu_nest_interface *ngpu_nest_interface_instance;
/* ----------------------------------------------------------------
 * Recordables map
 * ---------------------------------------------------------------- */

// Override the create() method with one call to
// DynamicRecordablesMap::insert() for each quantity to be recorded.
template <>
void
DynamicRecordablesMap< ngpu_nest_interface >::create( ngpu_nest_interface& host )
{
  // use standard names wherever you can for consistency!
  //insert( names::V_m, host.get_data_access_functor( ngpu_nest_interface::State_::V_M ) );

  //insert( names::w, host.get_data_access_functor( ngpu_nest_interface::State_::W ) );

  //host.insert_conductance_recordables();
}

/* ----------------------------------------------------------------
 * Default constructors defining default parameters and state
 * ---------------------------------------------------------------- */

ngpu_nest_interface::ngpu_nest_interface()
  : Node()
{
  recordablesMap_.create( *this );
  ngpu_nest_interface_instance = this;
}

ngpu_nest_interface::ngpu_nest_interface( const ngpu_nest_interface& n )
  : Node( n )
{
  // deve dare errore!!!
  recordablesMap_.create( *this );
  ngpu_nest_interface_instance = this;
}

ngpu_nest_interface::~ngpu_nest_interface()
{
}

/* ----------------------------------------------------------------
 * Node initialization functions
 * ---------------------------------------------------------------- */

void
ngpu_nest_interface::init_state_( const Node& proto )
{
  //const ngpu_nest_interface& pr = downcast< ngpu_nest_interface >( proto );
  //S_ = pr.S_;
}


/* ----------------------------------------------------------------
 * Default and copy constructor for node, and destructor
 * ---------------------------------------------------------------- */

/* ----------------------------------------------------------------
 * Node initialization functions
 * ---------------------------------------------------------------- */

void
ngpu_nest_interface::init_buffers_()
{
  for (uint i=0; i<spike_target_buffer_.size(); i++) {
    spike_target_buffer_[i].clear();   // includes resize
    //B_.step_ = Time::get_resolution().get_ms();
  }
}

void
ngpu_nest_interface::calibrate()
{
  //spike_target_buffer_.resize( n_receptors() );
  for (uint i=0; i<spike_target_buffer_.size(); i++) {
    spike_target_buffer_[i].clear();   // includes resize
    //B_.step_ = Time::get_resolution().get_ms();
  }

}

/* ----------------------------------------------------------------
 * Update and spike handling functions
 * ---------------------------------------------------------------- */
void
ngpu_nest_interface::update( Time const& origin, const long from, const long to )
{
  assert( to >= 0 && ( delay ) from < kernel().connection_manager.get_min_delay() );
  assert( from < to );
  assert( from >=0 );
  //std::cout << "update from: " << from << "  to: " << to   << "\n";

  for ( long lag = from; lag < to; ++lag ) // proceed by stepsize B_.step_
  {
    NeuronGPU_SimulationStep();
    /*
    if (lag<spike_target_buffer_.size()) {
      for (uint i=0; i<spike_target_buffer_[lag].size(); i++) {
	int image_id = spike_target_buffer_[lag][i];
	std::cout << "update image id: " << image_id << " lag:" << lag << "\n";
      }
    }
    */
    if (lag<(int)spike_target_buffer_.size()
	&& spike_target_buffer_[lag].size()>0) {
      NeuronGPU_PushSpikesToNodes(spike_target_buffer_[lag].size(),
				  spike_target_buffer_[lag].data());
      spike_target_buffer_[lag].clear();
    }
    int n_spikes;
    int *spike_node;
    int *spike_port;
    float *spike_height;
    NeuronGPU_GetExtNeuronInputSpikes(&n_spikes, &spike_node, &spike_port,
				      &spike_height, false);
    if (n_spikes>0) {
      //std::cout << "Received " << n_spikes << "input spikes from ngpu\n";
      //std::cout << "index\tnode\tport\theight\n";
      for (int i=0; i<n_spikes; i++) {
	//std::cout << i << "\t" << spike_node[i] << "\t" << spike_port[i]
	//	  << "\t" << spike_height[i] << "\n";
	int i_node_image = spike_node[i];
	int pos = (int)(std::find(node_image_id_.begin(), node_image_id_.end(),
				  i_node_image) - node_image_id_.begin());
        if(pos >= (int)node_image_id_.size()) {
	  std::cerr << "Unrecognized node image id\n";
           exit(0);
        }
	int nest_node_id = nest_node_id_[pos];
	//std::cout << "nest node id: " << nest_node_id << "\n";
	Node *nest_node = kernel().node_manager.get_node_or_proxy
			    (nest_node_id);
	SpikeEvent e;
	e.set_sender( *this );
	e.set_receiver(*nest_node);
	e.set_stamp( kernel().simulation_manager.get_slice_origin()
		     + Time::step( lag + 1 ) );
	e.set_weight(spike_height[i]);
	e.set_delay_steps(1);
	e.set_rport(spike_port[i]);
	e();
	//nest_node->handle(e);

      }
    }
    // log state data
    //B_.logger_.record_data( origin.get_steps() + lag );
  } // for-loop

}

port
ngpu_nest_interface::handles_test_event( SpikeEvent&, rport receptor_type )
{
  if ( receptor_type < 0 || receptor_type >= static_cast< port >( n_receptors() ) )
  {
    throw IncompatibleReceptorType( receptor_type, get_name(), "SpikeEvent" );
  }
  //P_.has_connections_ = true;
  return receptor_type;
}

void
ngpu_nest_interface::handle( SpikeEvent& e )
{
  if ( e.get_weight() < 0 )
  {
    throw BadProperty(
      "Synaptic weights for conductance-based multisynapse models "
      "must be positive." );
  }
  assert( ( e.get_rport() >= 0 ) && ( ( size_t ) e.get_rport() < n_receptors() ) );

  //std::cout << "Got spike w:" << e.get_weight() << " port:" << e.get_rport() << "\n";
  //B_.spikes_[ e.get_rport() - 1 ].add_value(
  //  e.get_rel_delivery_steps( kernel().simulation_manager.get_slice_origin() ), e.get_weight() * e.get_multiplicity() );
  int port = e.get_rport();
  int image_id = node_image_id_[port];
  int lag =
    e.get_rel_delivery_steps(kernel().simulation_manager.get_slice_origin())
    - 1;
  assert(lag>=0);
  //std::cout << "image id: " << image_id << " lag:" << lag << "\n";
  while ((int)spike_target_buffer_.size()<=lag) {
    spike_target_buffer_.push_back(std::vector<int>());
  }
  spike_target_buffer_[lag].push_back(image_id);
}

void
ngpu_nest_interface::set_status( const DictionaryDatum& d )
{
}

int ngpu_nest_interface::AddNode(int nest_node_id, int node_image_id)
{
  nest_node_id_.push_back(nest_node_id);
  node_image_id_.push_back(node_image_id);
  return (int)nest_node_id_.size() - 1;
}

  

} // namespace nest

extern "C" {

int ngpu_nest_Init()
{
  NeuronGPU_SetVerbosityLevel(0);
  NeuronGPU_SetSimTime(0.1);
  const Name& name="ngpu";
  int ngpu_model_index = nest::kernel().model_manager.register_node_model<nest::ngpu_nest_interface>(name);
  return ngpu_model_index;
}

int ngpu_nest_AddNode(int nest_node_id, int node_image_id)
{
  return nest::ngpu_nest_interface_instance->AddNode(nest_node_id,
						     node_image_id);
}

}
